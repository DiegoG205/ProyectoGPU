#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <random>

// CUDA
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>

#define PI 3.14159265358979323846

__device__ float smoothingKernel(float radius, float dist) {
  float volume = PI * std::pow(radius, 4) / 6;
  float value = max(0.0f, radius - dist);
  return (value * value / volume);
};

__device__ float smoothingKernelDerivative(float radius, float dist) {
  float scale = 12/(std::pow(radius, 4) * PI);
  float value = max(0.f, dist - radius);
  return value * scale;
};

__device__ float calculateDensity(int n, float3 pos, float3* positions, float radius) {
  
  float density = 0;
  const float mass = 1;

  for (int i = 0; i < n; i++) {
    
    float3 other = positions[i];

    float dx = pos.x - other.x;
    float dy = pos.y - other.y;
    float dz = pos.z - other.z;
    float dist = std::sqrt(dx*dx + dy*dy + dz*dz);

    float influence = smoothingKernel(radius, dist);

    density += influence * mass;
  }

  return density;
};

__device__ float densityToPressure(float density, float targetDensity, float pressureMultiplier) {
  float delta = density - targetDensity;
  return (delta * pressureMultiplier);
};

__device__ float3 calculatePressure(int n, float3 pos, float3* positions, float* densities, float radius, float trgDen, float pressMult) {

  float3 pressure = {0,0,0};
  float mass = 1;
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  for(int i = 0; i < n; i++) {
    
    if (i == index) continue;

    float3 other = positions[i];

    float dx = pos.x - other.x;
    float dy = pos.y - other.y;
    float dz = pos.z - other.z;
    float dist = std::sqrt(dx*dx + dy*dy + dz*dz + 0.01);
    float3 dir = {-dx/dist, -dy/dist, -dz/dist};

    float slope = smoothingKernelDerivative(radius, dist);

    float density = densities[i];
    float val = densityToPressure(density, trgDen, pressMult) * slope * mass / density;

    pressure.x += -dir.x * val;
    pressure.y += -dir.y * val;
    pressure.z += -dir.z * val;
  }

  return pressure;
}

__global__ void updateDensities(int n, float3 *posData, float *densities, float radius) {
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  densities[index] = calculateDensity(n, posData[index], posData, radius);
  printf("%d: %f\n", index, densities[index]);
};

// TODO: Revisar si es seguro eliminar velAux (cada particula accede solo a su velocidad => no hay datarace)
__global__ void fluid_kernel(int n, float3 *posData, float3* posAux, float3 *velData, float3 *velAux, float *densities, float dt, 
                             float radius, float trgDen, float pressMult) {

  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;


  float3 pos = posData[index];
  float3 vel = velData[index];

  // Gravity
  //vel.y -= 9.8 * dt;

  // Pressure force
  float3 pressure = calculatePressure(n, pos, posData, densities, radius, trgDen, pressMult);
  float3 pressureAcc = {pressure.x / densities[index], pressure.y / densities[index], pressure.z / densities[index]};

  // TODO: sumar la aceleracion, no asignarla directamente
  vel.x = pressureAcc.x * dt;
  vel.y = pressureAcc.y * dt;
  vel.z = pressureAcc.z * dt;

  // Border Collisions
  if (std::abs(pos.y + 0.05) >= 19.95) vel.y = -vel.y*0.7;
  if (std::abs(pos.x + 0.05) >= 19.95) vel.x = -vel.x*0.7;
  
  pos.x += vel.x * dt;
  pos.y += vel.y * dt;
  pos.z += vel.z * dt;

  posAux[index] = pos;
  velAux[index] = vel;

};

float calculateDensityHost(int n, float3 pos, float3* positions, float radius) {
  
  float density = 0;
  const float mass = 1;
  float volume = PI * std::pow(radius, 4) / 6;

  for (int i = 0; i < n; i++) {

    float3 other = positions[i];

    float dx = pos.x - other.x;
    float dy = pos.y - other.y;
    float dz = pos.z - other.z;
    float dist = std::sqrt(dx*dx + dy*dy + dz*dz);

    float value = max(0.0f, radius - dist);

    float influence = (value * value / volume);

    density += influence * mass;
  }

  return density;
};